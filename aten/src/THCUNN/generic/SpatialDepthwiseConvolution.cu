
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/SpatialDepthwiseConvolution.cu"
#else

void THNN_(SpatialDepthwiseConvolution_updateOutput)(
                  THCState *state,
                  THCTensor *input,
                  THCTensor *output,
                  THCTensor *weight,
                  THCTensor *bias,
                  int kW, int kH,
                  int dW, int dH,
                  int padW, int padH,
                  int dilationW, int dilationH)
{
  THCUNN_assertSameGPU(state, 3, input, output, weight);

  // Only handle 4D Input Tensors for now
  THAssert(!input->is_empty() && THCTensor_(nDimensionLegacyNoScalars)(state, input) == 4);
  THAssert(!weight->is_empty() && THCTensor_(nDimensionLegacyNoScalars)(state, weight) == 4);

  // We assume that the input and weight Tensors are shaped properly by
  // the caller, so we verify that here to some extent

  // Weight Tensor is shape (output_channels, 1, kH, kW)
  THAssert(weight->size(1) == 1);

  // Input Tensor is shape (N, input_channels, H, W)
  // We verify that the # of output_channels is a multiple of input_channels
  THAssert(weight->size(0) % input->size(1) == 0);

  // Bias has same # of channels as output
  if (bias) {
    THAssert(THTensor_sizeLegacyNoScalars(bias, 0) == weight->size(0));
  }

  input = THCTensor_(newContiguous)(state, input);
  weight = THCTensor_(newContiguous)(state, weight);
  bias = bias ? THCTensor_(newContiguous)(state, bias) : bias;

  // Following the behvaior of other THCUNN functions, we shape the output
  // Tensor ourselves

  int batchSize = input->size(0);
  int height = input->size(2);
  int width = input->size(3);
  int outputHeight = (height + 2 * padH - (dilationH * (kH - 1) + 1)) / dH + 1;
  int outputWidth = (width + 2 * padW - (dilationW * (kW - 1) + 1)) / dW + 1;
  int outputChannels = weight->size(0);

  THCTensor_(resize4d)(state, output, batchSize, outputChannels, outputHeight, outputWidth);

  // Create THCDeviceTensor
  // Kernel currently relies upon all the Tensors to be contiguous, but we made
  // them contiguous above
  THCDeviceTensor<scalar_t, 4> dInput = toDeviceTensor<scalar_t, 4>(state, input);
  THCDeviceTensor<scalar_t, 4> dWeight = toDeviceTensor<scalar_t, 4>(state, weight);
  THCDeviceTensor<scalar_t, 4> dOutput = toDeviceTensor<scalar_t, 4>(state, output);
  THCDeviceTensor<scalar_t, 1> dBias;
  if (bias) {
    dBias = toDeviceTensor<scalar_t, 1>(state, bias);
  }

  int inputChannels = input->size(1);
  int depthwiseMultiplier = outputChannels / inputChannels;

  // One thread per output value
  int n = THCTensor_(nElement)(state, output);
  int blocks = GET_BLOCKS(n);
  dim3 grid(blocks);
  dim3 block(CUDA_NUM_THREADS);
  if (kW == 3 && kH == 3) {
  spatialDepthwiseConvolutionUpdateOutput<scalar_t, accreal, unsigned int, 3><<<grid, block, 0, THCState_getCurrentStream(state)>>>(
    dInput, dOutput, dWeight, dBias, bias != NULL, n, outputChannels, depthwiseMultiplier,
    width, height, outputWidth, outputHeight,
    kW, kH, dW, dH, padW, padH, dilationW, dilationH);
  } else if (kW == 1 && kH == 1) {
  spatialDepthwiseConvolutionUpdateOutput<scalar_t, accreal, unsigned int, 1><<<grid, block, 0, THCState_getCurrentStream(state)>>>(
    dInput, dOutput, dWeight, dBias, bias != NULL, n, outputChannels, depthwiseMultiplier,
    width, height, outputWidth, outputHeight,
    kW, kH, dW, dH, padW, padH, dilationW, dilationH);
  } else {
  spatialDepthwiseConvolutionUpdateOutput<scalar_t, accreal, unsigned int, 0><<<grid, block, 0, THCState_getCurrentStream(state)>>>(
    dInput, dOutput, dWeight, dBias, bias != NULL, n, outputChannels, depthwiseMultiplier,
    width, height, outputWidth, outputHeight,
    kW, kH, dW, dH, padW, padH, dilationW, dilationH);
  }

  THCudaCheck(cudaGetLastError());

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, weight);
  if (bias) THCTensor_(free)(state, bias);
}

void THNN_(SpatialDepthwiseConvolution_updateGradInput)(
                  THCState *state,
                  THCTensor *input,
                  THCTensor *gradOutput,
                  THCTensor *gradInput,
                  THCTensor *weight,
                  int kW, int kH,
                  int dW, int dH,
                  int padW, int padH,
                  int dilationW, int dilationH)
{
  THCUNN_assertSameGPU(state, 3, gradOutput, gradInput, weight);

  // Only handle 4D Input Tensors for now
  THAssert(!input->is_empty() && THCTensor_(nDimensionLegacyNoScalars)(state, input) == 4);
  THAssert(!weight->is_empty() && THCTensor_(nDimensionLegacyNoScalars)(state, weight) == 4);
  THAssert(!gradOutput->is_empty() && THCTensor_(nDimensionLegacyNoScalars)(state, gradOutput) == 4);

  // Minimal shape checking, as above
  // Same # of elements in batch
  THAssert(input->size(0) == gradOutput->size(0));
  // Same # of filters as outputChannels
  THAssert(weight->size(0) == gradOutput->size(1));

  // Resize GradInput
  THCTensor_(resizeAs)(state, gradInput, input);

  int inputChannels = input->size(1);
  int height = input->size(2);
  int width = input->size(3);

  int outputChannels = gradOutput->size(1);
  int outputHeight = gradOutput->size(2);
  int outputWidth = gradOutput->size(3);

  int depthwiseMultiplier = outputChannels / inputChannels;

  THCDeviceTensor<scalar_t, 4> dGradOutput = toDeviceTensor<scalar_t, 4>(state, gradOutput);
  THCDeviceTensor<scalar_t, 4> dGradInput = toDeviceTensor<scalar_t, 4>(state, gradInput);
  THCDeviceTensor<scalar_t, 4> dWeight = toDeviceTensor<scalar_t, 4>(state, weight);

  // Kernel currently relies upon all the Tensors to be contiguous
  THAssert(dGradOutput.isContiguous());
  THAssert(dGradInput.isContiguous());
  THAssert(dWeight.isContiguous());

  // One thread per gradInput value
  int n = THCTensor_(nElement)(state, gradInput);
  int blocks = GET_BLOCKS(n);
  dim3 grid(blocks);
  dim3 block(CUDA_NUM_THREADS);
  if (kW == 3 && kH == 3)
    if (dW == 1 && dH == 1){
      spatialDepthwiseConvolutionUpdateGradInput<scalar_t, accreal, unsigned int, 3, 1><<<grid, block, 0, THCState_getCurrentStream(state)>>>(
      dGradOutput, dGradInput, dWeight, n, inputChannels, depthwiseMultiplier, outputChannels, width,
      height, outputWidth, outputHeight, kW, kH, dW, dH, padW, padH, dilationW, dilationH);
    } else if (dW == 2 && dH == 2) {
      spatialDepthwiseConvolutionUpdateGradInput<scalar_t, accreal, unsigned int, 3, 2><<<grid, block, 0, THCState_getCurrentStream(state)>>>(
      dGradOutput, dGradInput, dWeight, n, inputChannels, depthwiseMultiplier, outputChannels, width,
      height, outputWidth, outputHeight, kW, kH, dW, dH, padW, padH, dilationW, dilationH);
    } else {
      spatialDepthwiseConvolutionUpdateGradInput<scalar_t, accreal, unsigned int, 3, 0><<<grid, block, 0, THCState_getCurrentStream(state)>>>(
      dGradOutput, dGradInput, dWeight, n, inputChannels, depthwiseMultiplier, outputChannels, width,
      height, outputWidth, outputHeight, kW, kH, dW, dH, padW, padH, dilationW, dilationH);
    }
  else if (kW == 1 && kH == 1)
    if (dW == 1 && dH == 1){
      spatialDepthwiseConvolutionUpdateGradInput<scalar_t, accreal, unsigned int, 1, 1><<<grid, block, 0, THCState_getCurrentStream(state)>>>(
      dGradOutput, dGradInput, dWeight, n, inputChannels, depthwiseMultiplier, outputChannels, width,
      height, outputWidth, outputHeight, kW, kH, dW, dH, padW, padH, dilationW, dilationH);
    } else if (dW == 2 && dH == 2) {
      spatialDepthwiseConvolutionUpdateGradInput<scalar_t, accreal, unsigned int, 1, 2><<<grid, block, 0, THCState_getCurrentStream(state)>>>(
      dGradOutput, dGradInput, dWeight, n, inputChannels, depthwiseMultiplier, outputChannels, width,
      height, outputWidth, outputHeight, kW, kH, dW, dH, padW, padH, dilationW, dilationH);
    } else {
      spatialDepthwiseConvolutionUpdateGradInput<scalar_t, accreal, unsigned int, 1, 0><<<grid, block, 0, THCState_getCurrentStream(state)>>>(
      dGradOutput, dGradInput, dWeight, n, inputChannels, depthwiseMultiplier, outputChannels, width,
      height, outputWidth, outputHeight, kW, kH, dW, dH, padW, padH, dilationW, dilationH);
    }
  else
    if (dW == 1 && dH == 1){
      spatialDepthwiseConvolutionUpdateGradInput<scalar_t, accreal, unsigned int, 0, 1><<<grid, block, 0, THCState_getCurrentStream(state)>>>(
      dGradOutput, dGradInput, dWeight, n, inputChannels, depthwiseMultiplier, outputChannels, width,
      height, outputWidth, outputHeight, kW, kH, dW, dH, padW, padH, dilationW, dilationH);
    } else if (dW == 2 && dH == 2) {
      spatialDepthwiseConvolutionUpdateGradInput<scalar_t, accreal, unsigned int, 0, 2><<<grid, block, 0, THCState_getCurrentStream(state)>>>(
      dGradOutput, dGradInput, dWeight, n, inputChannels, depthwiseMultiplier, outputChannels, width,
      height, outputWidth, outputHeight, kW, kH, dW, dH, padW, padH, dilationW, dilationH);
    } else {
      spatialDepthwiseConvolutionUpdateGradInput<scalar_t, accreal, unsigned int, 0, 0><<<grid, block, 0, THCState_getCurrentStream(state)>>>(
      dGradOutput, dGradInput, dWeight, n, inputChannels, depthwiseMultiplier, outputChannels, width,
      height, outputWidth, outputHeight, kW, kH, dW, dH, padW, padH, dilationW, dilationH);
    }


  THCudaCheck(cudaGetLastError());
}

void THNN_(SpatialDepthwiseConvolution_accGradParameters)(
                  THCState *state,
                  THCTensor *input,
                  THCTensor *gradOutput,
                  THCTensor *gradWeight,
                  int kW, int kH,
                  int dW, int dH,
                  int padW, int padH,
                  int dilationW, int dilationH)
{
  THCUNN_assertSameGPU(state, 3, input, gradOutput, gradWeight);

  // Only handle 4D Input Tensors for now
  THAssert(!input->is_empty() && THCTensor_(nDimensionLegacyNoScalars)(state, input) == 4);
  THAssert(!gradOutput->is_empty() && THCTensor_(nDimensionLegacyNoScalars)(state, gradOutput) == 4);
  THAssert(!gradWeight->is_empty() && THCTensor_(nDimensionLegacyNoScalars)(state, gradWeight) == 4);

  // Minimal shape checking as above
  // Same # of elements in batch
  THAssert(input->size(0) == gradOutput->size(0));
  // Same # of filters as outputChannels
  THAssert(gradWeight->size(0) == gradOutput->size(1));

  int batchSize = input->size(0);
  int inputChannels = input->size(1);
  int height = input->size(2);
  int width = input->size(3);

  int outputChannels = gradOutput->size(1);
  int outputHeight = gradOutput->size(2);
  int outputWidth = gradOutput->size(3);

  int depthwiseMultiplier = outputChannels / inputChannels;

  THCDeviceTensor<scalar_t, 4> dGradOutput = toDeviceTensor<scalar_t, 4>(state, gradOutput);
  THCDeviceTensor<scalar_t, 4> dInput = toDeviceTensor<scalar_t, 4>(state, input);
  THCDeviceTensor<scalar_t, 4> dGradWeight = toDeviceTensor<scalar_t, 4>(state, gradWeight);

  // Kernel currently relies upon all the Tensors to be contiguous
  THAssert(dGradOutput.isContiguous());
  THAssert(dInput.isContiguous());
  THAssert(dGradWeight.isContiguous());

  // We parallelize so that each block computes a single value in gradWeight
  int blocks = outputChannels * kH * kW;


  // Make sure we have enough threads to perform the reduction, and use this number
  // to create the shared memory size for the reduction
  dim3 grid(blocks);
  dim3 block(getGradParamsNumThreads(batchSize));
  int smem = block.x * sizeof(accreal);

  spatialDepthwiseConvolutionAccGradParameters<scalar_t, accreal, unsigned int><<<grid, block, smem, THCState_getCurrentStream(state)>>>(
      dGradOutput, dInput, dGradWeight, batchSize, inputChannels, outputChannels, depthwiseMultiplier,
      width, height, outputWidth, outputHeight, kW, kH, dW, dH, padW, padH, dilationW, dilationH);

  THCudaCheck(cudaGetLastError());
}

void THNN_(SpatialDepthwiseConvolutionTBC_updateOutput)(
                  THCState *state,
                  THCTensor *input,
                  THCTensor *output,
                  THCTensor *weight,
                  THCTensor *bias,
                  int kW, int kH,
                  int dW, int dH,
                  int padW, int padH,
                  int dilationW, int dilationH)
{
  THCUNN_assertSameGPU(state, 3, input, output, weight);

  // Only handle 4D Input Tensors for now
  THAssert(!input->is_empty() && THCTensor_(nDimensionLegacyNoScalars)(state, input) == 4);
  THAssert(!weight->is_empty() && THCTensor_(nDimensionLegacyNoScalars)(state, weight) == 4);

  // We assume that the input and weight Tensors are shaped properly by
  // the caller, so we verify that here to some extent

  // Weight Tensor is shape (kH, kW, 1, output_channels)
  THAssert(weight->size(2) == 1);

  // Input Tensor is shape (H, W, N, input_channels)
  // We verify that the # of output_channels is a multiple of input_channels
  THAssert(weight->size(3) % input->size(3) == 0);

  // Bias has same # of channels as output
  if (bias) {
    THAssert(THTensor_sizeLegacyNoScalars(bias, 0) == weight->size(3));
  }

  input = THCTensor_(newContiguous)(state, input);
  weight = THCTensor_(newContiguous)(state, weight);
  bias = bias ? THCTensor_(newContiguous)(state, bias) : bias;

  // Following the behvaior of other THCUNN functions, we shape the output
  // Tensor ourselves

  int batchSize = input->size(2);
  int height = input->size(0);
  int width = input->size(1);
  int outputHeight = (height + 2 * padH - (dilationH * (kH - 1) + 1)) / dH + 1;
  int outputWidth = (width + 2 * padW - (dilationW * (kW - 1) + 1)) / dW + 1;
  int outputChannels = weight->size(3);

  THCTensor_(resize4d)(state, output, outputHeight, outputWidth, batchSize, outputChannels);

  // Create THCDeviceTensor
  // Kernel currently relies upon all the Tensors to be contiguous, but we made
  // them contiguous above
  THCDeviceTensor<scalar_t, 4> dInput = toDeviceTensor<scalar_t, 4>(state, input);
  THCDeviceTensor<scalar_t, 4> dWeight = toDeviceTensor<scalar_t, 4>(state, weight);
  THCDeviceTensor<scalar_t, 4> dOutput = toDeviceTensor<scalar_t, 4>(state, output);
  THCDeviceTensor<scalar_t, 1> dBias;
  if (bias) {
    dBias = toDeviceTensor<scalar_t, 1>(state, bias);
  }

  int inputChannels = input->size(3);
  int depthwiseMultiplier = outputChannels / inputChannels;

  // One thread per output value
  int n = THCTensor_(nElement)(state, output);
  int blocks = GET_BLOCKS(n);
  dim3 grid(blocks);
  dim3 block(CUDA_NUM_THREADS);
  if (kW == 3 && kH == 3) {
  spatialDepthwiseConvolutionTBCUpdateOutput<scalar_t, accreal, unsigned int, 3><<<grid, block, 0, THCState_getCurrentStream(state)>>>(
    dInput, dOutput, dWeight, dBias, bias != NULL, n, outputChannels, depthwiseMultiplier,
    width, height, outputWidth, outputHeight,
    kW, kH, dW, dH, padW, padH, dilationW, dilationH);
  } else if (kW == 1 && kH == 1) {
  spatialDepthwiseConvolutionTBCUpdateOutput<scalar_t, accreal, unsigned int, 1><<<grid, block, 0, THCState_getCurrentStream(state)>>>(
    dInput, dOutput, dWeight, dBias, bias != NULL, n, outputChannels, depthwiseMultiplier,
    width, height, outputWidth, outputHeight,
    kW, kH, dW, dH, padW, padH, dilationW, dilationH);
  } else {
  spatialDepthwiseConvolutionTBCUpdateOutput<scalar_t, accreal, unsigned int, 0><<<grid, block, 0, THCState_getCurrentStream(state)>>>(
    dInput, dOutput, dWeight, dBias, bias != NULL, n, outputChannels, depthwiseMultiplier,
    width, height, outputWidth, outputHeight,
    kW, kH, dW, dH, padW, padH, dilationW, dilationH);
  }

  THCudaCheck(cudaGetLastError());

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, weight);
  if (bias) THCTensor_(free)(state, bias);
}

void THNN_(SpatialDepthwiseConvolutionTBC_updateGradInput)(
                  THCState *state,
                  THCTensor *input,
                  THCTensor *gradOutput,
                  THCTensor *gradInput,
                  THCTensor *weight,
                  int kW, int kH,
                  int dW, int dH,
                  int padW, int padH,
                  int dilationW, int dilationH)
{
  THCUNN_assertSameGPU(state, 3, gradOutput, gradInput, weight);

  // Only handle 4D Input Tensors for now
  THAssert(!input->is_empty() && THCTensor_(nDimensionLegacyNoScalars)(state, input) == 4);
  THAssert(!weight->is_empty() && THCTensor_(nDimensionLegacyNoScalars)(state, weight) == 4);
  THAssert(!gradOutput->is_empty() && THCTensor_(nDimensionLegacyNoScalars)(state, gradOutput) == 4);

  // input :  H,  W, N, iC
  // output: oH, oW, N, oC
  // weight: kH, kW, 1, oC
  // bias  : oC

  // Minimal shape checking, as above
  // Same # of elements in batch
  THAssert(input->size(2) == gradOutput->size(2));
  // Same # of filters as outputChannels
  THAssert(weight->size(3) == gradOutput->size(3));

  // Resize GradInput
  THCTensor_(resizeAs)(state, gradInput, input);

  int inputChannels = input->size(3);
  int height = input->size(0);
  int width = input->size(1);

  int outputChannels = gradOutput->size(3);
  int outputHeight = gradOutput->size(0);
  int outputWidth = gradOutput->size(1);

  int depthwiseMultiplier = outputChannels / inputChannels;

  THCDeviceTensor<scalar_t, 4> dGradOutput = toDeviceTensor<scalar_t, 4>(state, gradOutput);
  THCDeviceTensor<scalar_t, 4> dGradInput = toDeviceTensor<scalar_t, 4>(state, gradInput);
  THCDeviceTensor<scalar_t, 4> dWeight = toDeviceTensor<scalar_t, 4>(state, weight);

  // Kernel currently relies upon all the Tensors to be contiguous
  THAssert(dGradOutput.isContiguous());
  THAssert(dGradInput.isContiguous());
  THAssert(dWeight.isContiguous());

  // One thread per gradInput value
  int n = THCTensor_(nElement)(state, gradInput);
  int blocks = GET_BLOCKS(n);
  dim3 grid(blocks);
  dim3 block(CUDA_NUM_THREADS);
  if (kW == 3 && kH == 3)
    if (dW == 1 && dH == 1){
      spatialDepthwiseConvolutionTBCUpdateGradInput<scalar_t, accreal, unsigned int, 3, 1><<<grid, block, 0, THCState_getCurrentStream(state)>>>(
      dGradOutput, dGradInput, dWeight, n, inputChannels, depthwiseMultiplier, outputChannels, width,
      height, outputWidth, outputHeight, kW, kH, dW, dH, padW, padH, dilationW, dilationH);
    } else if (dW == 2 && dH == 2) {
      spatialDepthwiseConvolutionTBCUpdateGradInput<scalar_t, accreal, unsigned int, 3, 2><<<grid, block, 0, THCState_getCurrentStream(state)>>>(
      dGradOutput, dGradInput, dWeight, n, inputChannels, depthwiseMultiplier, outputChannels, width,
      height, outputWidth, outputHeight, kW, kH, dW, dH, padW, padH, dilationW, dilationH);
    } else {
      spatialDepthwiseConvolutionTBCUpdateGradInput<scalar_t, accreal, unsigned int, 3, 0><<<grid, block, 0, THCState_getCurrentStream(state)>>>(
      dGradOutput, dGradInput, dWeight, n, inputChannels, depthwiseMultiplier, outputChannels, width,
      height, outputWidth, outputHeight, kW, kH, dW, dH, padW, padH, dilationW, dilationH);
    }
  else if (kW == 1 && kH == 1)
    if (dW == 1 && dH == 1){
      spatialDepthwiseConvolutionTBCUpdateGradInput<scalar_t, accreal, unsigned int, 1, 1><<<grid, block, 0, THCState_getCurrentStream(state)>>>(
      dGradOutput, dGradInput, dWeight, n, inputChannels, depthwiseMultiplier, outputChannels, width,
      height, outputWidth, outputHeight, kW, kH, dW, dH, padW, padH, dilationW, dilationH);
    } else if (dW == 2 && dH == 2) {
      spatialDepthwiseConvolutionTBCUpdateGradInput<scalar_t, accreal, unsigned int, 1, 2><<<grid, block, 0, THCState_getCurrentStream(state)>>>(
      dGradOutput, dGradInput, dWeight, n, inputChannels, depthwiseMultiplier, outputChannels, width,
      height, outputWidth, outputHeight, kW, kH, dW, dH, padW, padH, dilationW, dilationH);
    } else {
      spatialDepthwiseConvolutionTBCUpdateGradInput<scalar_t, accreal, unsigned int, 1, 0><<<grid, block, 0, THCState_getCurrentStream(state)>>>(
      dGradOutput, dGradInput, dWeight, n, inputChannels, depthwiseMultiplier, outputChannels, width,
      height, outputWidth, outputHeight, kW, kH, dW, dH, padW, padH, dilationW, dilationH);
    }
  else
    if (dW == 1 && dH == 1){
      spatialDepthwiseConvolutionTBCUpdateGradInput<scalar_t, accreal, unsigned int, 0, 1><<<grid, block, 0, THCState_getCurrentStream(state)>>>(
      dGradOutput, dGradInput, dWeight, n, inputChannels, depthwiseMultiplier, outputChannels, width,
      height, outputWidth, outputHeight, kW, kH, dW, dH, padW, padH, dilationW, dilationH);
    } else if (dW == 2 && dH == 2) {
      spatialDepthwiseConvolutionTBCUpdateGradInput<scalar_t, accreal, unsigned int, 0, 2><<<grid, block, 0, THCState_getCurrentStream(state)>>>(
      dGradOutput, dGradInput, dWeight, n, inputChannels, depthwiseMultiplier, outputChannels, width,
      height, outputWidth, outputHeight, kW, kH, dW, dH, padW, padH, dilationW, dilationH);
    } else {
      spatialDepthwiseConvolutionTBCUpdateGradInput<scalar_t, accreal, unsigned int, 0, 0><<<grid, block, 0, THCState_getCurrentStream(state)>>>(
      dGradOutput, dGradInput, dWeight, n, inputChannels, depthwiseMultiplier, outputChannels, width,
      height, outputWidth, outputHeight, kW, kH, dW, dH, padW, padH, dilationW, dilationH);
    }


  THCudaCheck(cudaGetLastError());
}

void THNN_(SpatialDepthwiseConvolutionTBC_accGradParameters)(
                  THCState *state,
                  THCTensor *input,
                  THCTensor *gradOutput,
                  THCTensor *gradWeight,
                  int kW, int kH,
                  int dW, int dH,
                  int padW, int padH,
                  int dilationW, int dilationH)
{
  THCUNN_assertSameGPU(state, 3, input, gradOutput, gradWeight);

  // input :  H,  W, N, iC
  // output: oH, oW, N, oC
  // weight: kH, kW, 1, oC
  // bias  : oC

  // Only handle 4D Input Tensors for now
  THAssert(!input->is_empty() && THCTensor_(nDimensionLegacyNoScalars)(state, input) == 4);
  THAssert(!gradOutput->is_empty() && THCTensor_(nDimensionLegacyNoScalars)(state, gradOutput) == 4);
  THAssert(!gradWeight->is_empty() && THCTensor_(nDimensionLegacyNoScalars)(state, gradWeight) == 4);

  // Minimal shape checking as above
  // Same # of elements in batch
  THAssert(input->size(2) == gradOutput->size(2));
  // Same # of filters as outputChannels
  THAssert(gradWeight->size(3) == gradOutput->size(3));

  int batchSize = input->size(2);
  int inputChannels = input->size(3);
  int height = input->size(0);
  int width = input->size(1);

  int outputChannels = gradOutput->size(3);
  int outputHeight = gradOutput->size(0);
  int outputWidth = gradOutput->size(1);

  int depthwiseMultiplier = outputChannels / inputChannels;

  THCDeviceTensor<scalar_t, 4> dGradOutput = toDeviceTensor<scalar_t, 4>(state, gradOutput);
  THCDeviceTensor<scalar_t, 4> dInput = toDeviceTensor<scalar_t, 4>(state, input);
  THCDeviceTensor<scalar_t, 4> dGradWeight = toDeviceTensor<scalar_t, 4>(state, gradWeight);

  // Kernel currently relies upon all the Tensors to be contiguous
  THAssert(dGradOutput.isContiguous());
  THAssert(dInput.isContiguous());
  THAssert(dGradWeight.isContiguous());

  // One thread per gradWeight value
  int n = THCTensor_(nElement)(state, gradWeight);
  int blocks = GET_BLOCKS(n);
  dim3 grid(blocks);
  dim3 block(CUDA_NUM_THREADS);

  spatialDepthwiseConvolutionTBCAccGradParameters<scalar_t, accreal, unsigned int><<<grid, block, 0, THCState_getCurrentStream(state)>>>(
      dGradOutput, dInput, dGradWeight, batchSize, inputChannels, outputChannels, depthwiseMultiplier,
      width, height, outputWidth, outputHeight, kW, kH, dW, dH, padW, padH, dilationW, dilationH);

  THCudaCheck(cudaGetLastError());
}

#endif
